#include "hip/hip_runtime.h"
﻿#pragma once
#include <DECODE.h>
#include <hip/hip_runtime.h>
#include <>
#include <iostream>
#include <numeric>

//#define SCORE_MAX 140 //
#define PEPTIDE_LENGTH 12
__constant__ float g_AAMap[21 * 21];

int SCORE_MAX = 10;
char AAMap[21 * 21];

size_t task = 0;
size_t task_target = 0;
size_t samplesize = 0;

unsigned int Cores;
string scoretableFP = {};
//#define FloatSHIFT 100000.0
#define FloatSHIFT 1000.0
#define FloatSHIFT_INT 1000.0
#define FloatSHIFT_Ratio 1000.0

#define CUDA_SAFE_CALL(func) \
do { \
     hipError_t err = (func); \
     if (err != hipSuccess) { \
         fprintf(stderr, "[Error] %s (error code: %d) at %s line %d\n", hipGetErrorString(err), err, __FILE__, __LINE__); \
         exit(err); \
     } \
} while(0)

double P_TH[12] = { 1,0.05,0.01,0.005,0.001,0.0005,0.0001, 0.00005, 0.00001, 0.000005, 0.000001, 0.0000001 };


struct Option {
	string FPr = "./";
	string FPw = "./";
	string P_map = "./";
	string AAtable;
	string Target;
	string Start = "0";
	int threads = 1;
	int PepLimit = 0;
	int Q_Export = 0;
	int P_Export = 0;
	string Plistexport;
	int Calc = 0;               // 0: read of peptide, 1;kind of peptide
	int Distance_function = 0;  // 0:KL, 1:L1, 2:L2, 3:
	double TH = 0.05*FloatSHIFT;
} OPs;

struct PV {
	// vector<vector<float>> scorelist;
	string *filename;
	float *scorelist;  // P value
	char *Seq;         // Protein seq
	int *Seqlength;
	int scoresize = 0;
	int targetsize = 0;
	size_t totalAA = 0;
	size_t totalread = 0;
};

struct Pep_data {
	char *pep;
	float *read;
	int totalPeps = 0;  // kinds of pep
	int peplength = 0;  // kinds of pep x peplength
};

struct mems {
	//host
	size_t GPU_mem;
	size_t Host_mem;
	float *h_distance;
	float *h_AAratio;
	float *h_Scoreratio;
	float *h_read_TH;
	float *h_result;
	double *h_Read_Ratio;

	//device
	char *d_pep, *d_pro, *d_AAmap;
	float *d_AAratio;
	float *d_P_list;
	float *d_Scoreratio;
	float *d_distance;
	float *d_read;
	float *d_read_TH;
};


vector<vector<vector<string>>> queue;
vector<bool> queflag;
inline char S2C(char src) {
	switch (src) {
	case 'A':
		return 0;
	case 'R':
		return 1;
	case 'N':
		return 2;
	case 'D':
		return 3;
		break;
	case 'C':
		return 4;
		break;
	case 'Q':
		return 5;
		break;
	case 'E':
		return 6;
		break;
	case 'G':
		return 7;
		break;
	case 'H':
		return 8;
		break;
	case 'I':
		return 9;
		break;
	case 'L':
		return 10;
		break;
	case 'K':
		return 11;
		break;
	case 'M':
		return 12;
		break;
	case 'F':
		return 13;
		break;
	case 'P':
		return 14;
		break;
	case 'S':
		return 15;
		break;
	case 'T':
		return 16;
		break;
	case 'W':
		return 17;
		break;
	case 'Y':
		return 18;
		break;
	case 'V':
		return 19;
		break;
	case '*':
		return 20;
		break;
	default:
		return 20;
		break;
	}
	return 20;
}
inline char *AA2Chr2(string &src) {
	int length = src.length();
	char *dst = new char[length + 1];
	for (auto i = 0; i < length; ++i) {
		switch (src[i]) {
		case 'A':
			dst[i] = 0;
			break;
		case 'R':
			dst[i] = 1;
			break;
		case 'N':
			dst[i] = 2;
			break;
		case 'D':
			dst[i] = 3;
			break;
		case 'C':
			dst[i] = 4;
			break;
		case 'Q':
			dst[i] = 5;
			break;
		case 'E':
			dst[i] = 6;
			break;
		case 'G':
			dst[i] = 7;
			break;
		case 'H':
			dst[i] = 8;
			break;
		case 'I':
			dst[i] = 9;
			break;
		case 'L':
			dst[i] = 10;
			break;
		case 'K':
			dst[i] = 11;
			break;
		case 'M':
			dst[i] = 12;
			break;
		case 'F':
			dst[i] = 13;
			break;
		case 'P':
			dst[i] = 14;
			break;
		case 'S':
			dst[i] = 15;
			break;
		case 'T':
			dst[i] = 16;
			break;
		case 'W':
			dst[i] = 17;
			break;
		case 'Y':
			dst[i] = 18;
			break;
		case 'V':
			dst[i] = 19;
			break;
		case '*':
			dst[i] = 20;
			break;
		default:
			dst[i] = 20;
			break;
		}
	}
	dst[length] = '\0';
	return dst;
}
inline char *AA2Chr_pep2(char src[PEPTIDE_LENGTH + 1]) {
	char *dst = new char[PEPTIDE_LENGTH + 1];
	for (int i = 0; i < PEPTIDE_LENGTH + 1; ++i) {
		dst[i] = '\0';
	}
	for (auto i = 0; i < PEPTIDE_LENGTH; ++i) {
		switch (src[i]) {
		case 'A':
			dst[i] = 0;
			break;
		case 'R':
			dst[i] = 1;
			break;
		case 'N':
			dst[i] = 2;
			break;
		case 'D':
			dst[i] = 3;
			break;
		case 'C':
			dst[i] = 4;
			break;
		case 'Q':
			dst[i] = 5;
			break;
		case 'E':
			dst[i] = 6;
			break;
		case 'G':
			dst[i] = 7;
			break;
		case 'H':
			dst[i] = 8;
			break;
		case 'I':
			dst[i] = 9;
			break;
		case 'L':
			dst[i] = 10;
			break;
		case 'K':
			dst[i] = 11;
			break;
		case 'M':
			dst[i] = 12;
			break;
		case 'F':
			dst[i] = 13;
			break;
		case 'P':
			dst[i] = 14;
			break;
		case 'S':
			dst[i] = 15;
			break;
		case 'T':
			dst[i] = 16;
			break;
		case 'W':
			dst[i] = 17;
			break;
		case 'Y':
			dst[i] = 18;
			break;
		case 'V':
			dst[i] = 19;
			break;
		case '*':
			dst[i] = 20;
			break;
		default:
			dst[i] = 20;
			break;
		}
	}
	dst[12] = '\0';
	return dst;
}
Pep_data Import_Peptide(const string &FP) {
	cout << "Import_Peptide:" << FP << endl;
	char buf[256];
	FILE *stream;
	fopen_s(&stream, FP.c_str(), "r");
	vector<vector<string>> v;
	while (fgets(buf, 255, stream) != NULL) {
		vector<string> vbuf = split_comma(buf);
		if (stoi(vbuf[1]) < OPs.PepLimit)
			break;
		else {
			if (vbuf[0].length() == PEPTIDE_LENGTH)
				v.push_back(vbuf);
		}
	}
	fclose(stream);
	Pep_data data;
	char *pep = new char[v.size()*PEPTIDE_LENGTH];
	float *read = new float[v.size()];
	char *pP = pep;
	float *pR = read;
	size_t maxread = 1;
	maxread = stoi(v[0][1]);
	for (int i = 0; i < v.size(); ++i) {
		for (int j = 0; j < v[i][0].length(); ++j, ++pP) {
			*pP = S2C(v[i][0][j]);
		}
		if (stof(v[i][1]) <= maxread && stof(v[i][1]) > 0)
			read[i] = stof(v[i][1]);
		else
			read[i] = 1;
	}
	data.pep = pep;
	data.peplength = v.size()*PEPTIDE_LENGTH;
	data.read = read;
	data.totalPeps = v.size();
	return data;
}

inline void CalcDistance_KL(float *Q, const PV &P, float *d) {
	// Distance calculation by Kullback–Leibler divergence

	int slen = P.scoresize;
	int targetsize = P.targetsize;
	int s = 0;
	int i = 0;
	int j = 0;
	float *pQ = Q;
	float *pP = P.scorelist;
	float *pd = d;
	//float Plimit = 1 / P.totalread;
	double Plimit = (1.0 / (double)P.totalread) / FloatSHIFT;

	for (; i < targetsize; ++i) {
		for (j = 0; j < P.Seqlength[i]; ++j, ++pd) {
			for (s = 0; s < slen; ++s, ++pQ, ++pP) {
				if (j < P.Seqlength[i] - 12) {
					*pd += (double)*pP / FloatSHIFT * log((double)*pP / FloatSHIFT / ((double)*pQ / FloatSHIFT));
					/*
					if (*pQ / FloatSHIFT >= Plimit) {
						if (*pP / FloatSHIFT < Plimit && *pQ>0) {
							*pd += abs(Plimit * log(Plimit / (double)*pQ / FloatSHIFT));
						}
						else {
							// cout << abs(log(*fPs / *fQ)) << endl;
							*pd += abs((double)*pP / FloatSHIFT * log((double)*pP / FloatSHIFT / ((double)*pQ / FloatSHIFT)));
						}
					}
					*/
				}
			}
		}
	}
}
inline void CalcDistance_reverseKL(float *Q, const PV &P, float *d) {
	// Distance calculation by reverse Kullback–Leibler divergence

	int slen = P.scoresize;
	int targetsize = P.targetsize;
	int s = 0;
	int i = 0;
	int j = 0;
	float *pQ = Q;
	float *pP = P.scorelist;
	float *pd = d;
	//float Plimit = 1 / P.totalread;
	double Plimit = (float)(1.0 / (double)P.totalread) / FloatSHIFT;

	for (; i < targetsize; ++i) {
		for (j = 0; j < P.Seqlength[i]; ++j, ++pd) {
			for (s = 0; s < slen; ++s, ++pQ, ++pP) {
				if (j < P.Seqlength[i] - 12) {
					if (*pQ / FloatSHIFT >= Plimit) {
						if (*pP / FloatSHIFT < Plimit && *pQ>0) {
							*pd += abs((double)*pQ / FloatSHIFT * log((double)*pQ / FloatSHIFT / Plimit));
						}
						else {
							// cout << abs(log(*fPs / *fQ)) << endl;
							*pd += abs((double)*pQ / FloatSHIFT * log((double)*pQ / FloatSHIFT / ((double)*pP / FloatSHIFT)));
						}
					}
				}
			}
		}
	}
}
inline void CalcDistance_Ratio(float *Q, const PV &P, float *d) {
	// Distance calculation by log(P/Q) 
	int slen = P.scoresize;
	int targetsize = P.targetsize;
	int s = 0;
	int i = 0;
	int j = 0;
	float *pQ = Q;
	float *pP = P.scorelist;
	float *pd = d;
	//float Plimit = 1 / P.totalread;
	double Plimit = (float)(1.0 / (double)P.totalread) / FloatSHIFT;

	for (; i < targetsize; ++i) {
		for (j = 0; j < P.Seqlength[i]; ++j, ++pd) {
			for (s = 0; s < slen; ++s, ++pQ, ++pP) {
				if (j < P.Seqlength[i] - 12) {
					if (*pQ / FloatSHIFT >= Plimit) {
						if (*pP / FloatSHIFT < Plimit && *pQ>0) {
							*pd += (double)*pQ / FloatSHIFT / Plimit;
						}
						else {
							*pd += ((double)*pQ / FloatSHIFT) / ((double)*pP / FloatSHIFT);
						}
					}
				}
			}
		}
	}
}
inline void CalcDistance_Ratio_log(float *Q, const PV &P, float *d) {
	// Distance calculation by log(P/Q) 
	int slen = P.scoresize;
	int targetsize = P.targetsize;
	int s = 0;
	int i = 0;
	int j = 0;
	float *pQ = Q;
	float *pP = P.scorelist;
	float *pd = d;
	//float Plimit = 1 / P.totalread;
	double Plimit = (float)(1.0 / (double)P.totalread) / FloatSHIFT;

	for (; i < targetsize; ++i) {
		for (j = 0; j < P.Seqlength[i]; ++j, ++pd) {
			for (s = 0; s < slen; ++s, ++pQ, ++pP) {
				if (j < P.Seqlength[i] - 12) {
					if (*pQ / FloatSHIFT >= Plimit) {
						if (*pP / FloatSHIFT < Plimit && *pQ>0) {
							*pd += abs(log((double)*pQ / FloatSHIFT / Plimit));
						}
						else {
							*pd += abs(log((double)*pQ / FloatSHIFT / ((double)*pP / FloatSHIFT)));
						}
					}
				}
			}
		}
	}
}
inline void CalcDistance_L1(float *Q, const PV &P, float *d) {
	// Distance calculation by log(P/Q) 
	int slen = P.scoresize;
	int targetsize = P.targetsize;
	int s = 0;
	int i = 0;
	int j = 0;
	int count = 0;
	float *pQ = Q;
	float *pP = P.scorelist;
	float *pd = d;
	//float Plimit = 1 / P.totalread;
	double Plimit = (float)(1.0 / (double)P.totalread) / FloatSHIFT;

	for (; i < targetsize; ++i) {
		for (j = 0; j < P.Seqlength[i]; ++j, ++pd) {
			for (count = 0, s = 0; s < slen; ++s, ++pQ, ++pP) {
				if (j < P.Seqlength[i] - 12) {
					if (*pQ / FloatSHIFT >= Plimit) {
						if (*pP / FloatSHIFT < Plimit && *pQ>0) {
							*pd += abs((double)*pQ / FloatSHIFT - Plimit);
						}
						else {
							*pd += abs((double)*pQ / FloatSHIFT - (double)*pP / FloatSHIFT);
						}
						count++;
					}

				}
			}
			if (j < P.Seqlength[i] - 12) {
				*pd = *pd / count;
			}
		}
	}

}
inline void CalcDistance_L2(float *Q, const PV &P, float *d) {
	// Distance calculation by log(P/Q) 
	int slen = P.scoresize;
	int targetsize = P.targetsize;
	int s = 0;
	int i = 0;
	int j = 0;
	int count = 0;
	float *pQ = Q;
	float *pP = P.scorelist;
	float *pd = d;
	//float Plimit = 1 / P.totalread;
	double Plimit = (float)(1.0 / (double)P.totalread) / FloatSHIFT;

	for (; i < targetsize; ++i) {
		for (j = 0; j < P.Seqlength[i]; ++j, ++pd) {
			for (count = 0, s = 0; s < slen; ++s, ++pQ, ++pP) {
				if (j < P.Seqlength[i] - 12) {
					if (*pQ / FloatSHIFT >= Plimit) {
						if (*pP / FloatSHIFT < Plimit && *pQ>0) {
							*pd += pow(((double)*pQ / FloatSHIFT - Plimit), 2);
						}
						else {
							*pd += pow(((double)*pQ / FloatSHIFT - (double)*pP / FloatSHIFT), 2);
						}
						count++;
					}
				}
			}
			if (j < P.Seqlength[i] - 12) {
				*pd = *pd / count;
			}
		}
	}
}

inline void CalcDistance_Pearson(float *Q, const PV &P, float *d) {

	// Distance calculation by Pearson X2
	int slen = P.scoresize;
	int targetsize = P.targetsize;
	int s = 0;
	int i = 0;
	int j = 0;
	float *pQ = Q;
	float *pP = P.scorelist;
	float *pd = d;
	//float Plimit = 1 / P.totalread;
	double Plimit = (float)(1.0 / (double)P.totalread) / FloatSHIFT;

	for (; i < targetsize; ++i) {
		for (j = 0; j < P.Seqlength[i]; ++j, ++pd) {
			*pd = 0;
			for (s = 0; s < slen; ++s, ++pQ, ++pP) {
				if (j < P.Seqlength[i] - 12) {
					if (*pQ / FloatSHIFT >= Plimit) {

						if (*pP / FloatSHIFT < Plimit && *pQ>0) {
							*pd += pow(((double)*pQ / FloatSHIFT - Plimit), 2) / Plimit;
						}
						else {
							if (*pP > 0)
								*pd += pow(((double)*pQ / FloatSHIFT - (double)*pP / FloatSHIFT), 2) / ((double)*pP / FloatSHIFT);
						}
					}
				}
			}
		}
	}
}
inline void CalcDistance_PearsonX3(float *Q, const PV &P, float *d) {

	// Distance calculation by Pearson X2
	int slen = P.scoresize;
	int targetsize = P.targetsize;
	int s = 0;
	int i = 0;
	int j = 0;
	float *pQ = Q;
	float *pP = P.scorelist;
	float *pd = d;
	//float Plimit = 1 / P.totalread;
	double Plimit = (float)(1.0 / (double)P.totalread) / FloatSHIFT;

	for (; i < targetsize; ++i) {
		for (j = 0; j < P.Seqlength[i]; ++j, ++pd) {
			*pd = 0;
			for (s = 0; s < slen; ++s, ++pQ, ++pP) {
				if (j < P.Seqlength[i] - 12) {
					if (*pQ / FloatSHIFT >= Plimit) {

						if (*pP / FloatSHIFT < Plimit && *pQ>0) {
							*pd += pow(((double)*pQ / FloatSHIFT - Plimit), 3) / Plimit;
						}
						else {
							if (*pP > 0)
								*pd += pow(((double)*pQ / FloatSHIFT - (double)*pP / FloatSHIFT), 3) / ((double)*pP / FloatSHIFT);
						}
					}
				}
			}
		}
	}
}
inline void CalcDistance_SquaredHelinger(float *Q, const PV &P, float *d) {

	// Distance calculation by SquaredHelinger
	int slen = P.scoresize;
	int targetsize = P.targetsize;
	int s = 0;
	int i = 0;
	int j = 0;
	float *pQ = Q;
	float *pP = P.scorelist;
	float *pd = d;
	double Plimit = (float)(1.0 / (double)P.totalread) / FloatSHIFT;

	for (; i < targetsize; ++i) {
		for (j = 0; j < P.Seqlength[i]; ++j, ++pd) {
			for (s = 0; s < slen; ++s, ++pQ, ++pP) {
				if (j < P.Seqlength[i] - 12) {
					if (*pQ / FloatSHIFT >= Plimit) {
						if (*pP / FloatSHIFT < Plimit && *pQ>0) {
							*pd += pow((sqrt(Plimit) - sqrt((double)*pQ / FloatSHIFT)), 2);
						}
						else {
							*pd += pow((sqrt((double)*pP / FloatSHIFT) - sqrt((double)*pQ / FloatSHIFT)), 2);
						}
					}
				}
			}
		}
	}
}
inline void CalcDistance_JensonShannon(float *Q, const PV &P, float *d) {

	// Distance calculation by Jenson-Shannon
	int slen = P.scoresize;
	int targetsize = P.targetsize;
	int s = 0;
	int i = 0;
	int j = 0;
	float *pQ = Q;
	float *pP = P.scorelist;
	float *pd = d;
	double Plimit = (float)(1.0 / (double)P.totalread) / FloatSHIFT;

	for (; i < targetsize; ++i) {
		for (j = 0; j < P.Seqlength[i]; ++j, ++pd) {
			for (s = 0; s < slen; ++s, ++pQ, ++pP) {
				if (j < P.Seqlength[i] - 12) {
					if (*pQ >= Plimit) {
						if (*pP < Plimit && *pQ>0) {
							*pd += Plimit * (log((2 * Plimit) / ((Plimit) * ((double)*pQ / FloatSHIFT)))) + (double)*pQ / FloatSHIFT * (log(2 * ((double)*pQ / FloatSHIFT) / (((Plimit) * ((double)*pQ) / FloatSHIFT))));
						}
						else {
							*pd += (double)*pP / FloatSHIFT * (log((2 * (double)*pP / FloatSHIFT) / (((double)*pP / FloatSHIFT) * ((double)*pQ / FloatSHIFT)))) + (double)*pQ / FloatSHIFT * (log(2 * ((double)*pQ / FloatSHIFT) / ((((double)*pP / FloatSHIFT) * ((double)*pQ) / FloatSHIFT))));
						}
					}
				}
			}
			if (j < P.Seqlength[i] - 12) {
				*pd = *pd / 2;
			}
		}
	}
}

inline void CalcDistance(float *Q, const PV &P, float *d) {
	if (OPs.Distance_function == 0)
		CalcDistance_KL(Q, P, d);
	else if (OPs.Distance_function == 1)
		CalcDistance_Ratio(Q, P, d);
	else if (OPs.Distance_function == 2)
		CalcDistance_reverseKL(Q, P, d);
	else if (OPs.Distance_function == 3)
		CalcDistance_Pearson(Q, P, d);
	else if (OPs.Distance_function == 4)
		CalcDistance_SquaredHelinger(Q, P, d);
	else if (OPs.Distance_function == 5)
		CalcDistance_JensonShannon(Q, P, d);
	else if (OPs.Distance_function == 6)
		CalcDistance_Ratio_log(Q, P, d);
	else if (OPs.Distance_function == 7)
		CalcDistance_L1(Q, P, d);
	else if (OPs.Distance_function == 8)
		CalcDistance_L2(Q, P, d);
	else if (OPs.Distance_function == 30)
		CalcDistance_PearsonX3(Q, P, d);
}

inline void CalcReadRatio(float *Q, const PV &P, double *d) {
	int slen = P.scoresize;
	int targetsize = P.targetsize;
	int s = 0;
	int i = 0;
	int j = 0;
	float *pQ = Q;
	float *pP = P.scorelist;
	double *pd = d;


	for (; i < targetsize; ++i) {
		for (j = 0; j < P.Seqlength[i]; ++j, pd += 12) {
			*pd = 0;
			for (s = 0; s < slen; ++s, ++pQ, ++pP) {
				if (j < P.Seqlength[i] - 12) {
					for (int p = 0; p < 12; ++p) {
						if (*pP / FloatSHIFT >= P_TH[p]) {
							*(pd + p) = *pQ;
						}
					}
				}
			}
		}
	}


}
void Export_Q2(float *Q, const PV &P, const string &dir, const string &barcode) {
	// cout << "\rExport Q: " << dir + "\\Qlist\\" << barcode;
	Directory_check(dir + "\\Qlist2\\" + barcode);
	int targetsize = P.targetsize;
	int slen = P.scoresize;
	float *pQ = Q;
	for (int i = 0; i < targetsize; ++i) {
		// cout << "\rExport P : " << dir + "/Plist/" + P[i].filename + ".csv"
		FILE *stream;
		fopen_s(&stream, (dir + "\\Qlist2\\" + barcode + "\\" + P.filename[i] + ".csv").c_str(), "w");
		for (int j = 0; j < P.Seqlength[i]; ++j) {
			for (int k = 0; k < slen; ++k, ++pQ) {
				fprintf(stream, "%f,", *pQ);
			}
			fprintf(stream, "\n");
		}
		fclose(stream);
	}
}

void Export_Q(float *Q, const PV &P, const string &dir, const string &barcode) {
	// cout << "\rExport Q: " << dir + "\\Qlist\\" << barcode;
	Directory_check(dir + "\\Qlist\\" + barcode);
	int targetsize = P.targetsize;
	int slen = P.scoresize;
	float *pQ = Q;
	for (int i = 0; i < targetsize; ++i) {
		// cout << "\rExport P : " << dir + "/Plist/" + P[i].filename + ".csv"
		FILE *stream;
		fopen_s(&stream, (dir + "\\Qlist\\" + barcode + "\\" + P.filename[i] + ".csv").c_str(), "w");
		for (int j = 0; j < P.Seqlength[i]; ++j) {
			for (int k = 0; k < slen; ++k, ++pQ) {
				fprintf(stream, "%f,", *pQ);
			}
			fprintf(stream, "\n");
		}
		fclose(stream);
	}
}
void Export_P(const PV &P, const string &dir) {

	Directory_check(dir + "\\Plist");
	int targetsize = P.targetsize;
	int slen = P.scoresize;
	float *pP = P.scorelist;
	for (int i = 0; i < targetsize; ++i) {
		// cout << "\rExport P : " << dir + "/Plist/" + P[i].filename + ".csv"
		FILE *stream;
		fopen_s(&stream, (dir + "\\Plist\\" + P.filename[i] + ".csv").c_str(), "w");
		for (int j = 0; j < P.Seqlength[i]; ++j) {
			for (int k = 0; k < slen; ++k, ++pP) {
				fprintf(stream, "%f,", *pP);
			}
			fprintf(stream, "\n");
		}
		fclose(stream);
	}
}
inline void Export_result_header(const string &export_d, const string &fname) {
	Directory_check(export_d);
	string FP = export_d + "/" + fname + ".csv";
	FILE *stream;
	fopen_s(&stream, (FP).c_str(), "w");
	if (stream == NULL) {
		printf("%s file not open!\n", (FP).c_str());
	}
	else {
		// make header line
		// printf("Export: %s\n", (FP).c_str());
		fprintf(stream, ",pos,AA,read,distance");
		for (int n = 0; n < 21; ++n) {
			fprintf(stream, ",%c", AAlist[n]);
		}
		fprintf(stream, "\n");
		fclose(stream);
	}
}
inline void Export_result_bin(float *D, float *AA, u_int *read, const string &export_d, const string barcode, const PV &P) {
	Directory_check(export_d);

	string FP = export_d + "\\" + barcode + "_" + to_string(P.totalAA) + ".bin";

	FILE *stream;

	fopen_s(&stream, (FP).c_str(), "a");
	if (stream == NULL) {
		printf("%s file not open!\n", (FP).c_str());
	}
	else {


		char *buf = new char[1024 * 1024];
		setvbuf(stream, buf, _IOFBF, 512 * 512);
		int len = P.totalAA;
		u_int tlen = 0;
		u_int tsize = P.targetsize;
		string ID = "";
		int pos = 0;
		char *pAA = P.Seq;
		float *pD = D;
		u_int *pR = read;
		float* pRatio = AA;
		int i = 0; int j = 0; int k = 0;
		for (i = 0; i < tsize; ++i) {
			tlen = P.Seqlength[i];
			ID = P.filename[i].substr(1, P.filename[i].length() - 4);
			for (j = 0; j < tlen; ++j, ++pAA, ++pD, ++pR, ++pos) {
				fwrite(ID.c_str(), 1, ID.length(), stream);
				fwrite(&j, sizeof(int), 1, stream);
				fwrite(&AAlist[P.Seq[pos]], 1, 1, stream);
				fwrite(pR, sizeof(float), 1, stream);
				fwrite(pD, sizeof(float), 1, stream);

				for (k = 0; k < 21; ++k, ++pRatio) {
					fwrite(pRatio, sizeof(float), 1, stream);
				}
			}
		}

		fclose(stream);
		delete[] buf;
	}
	/*
	ofstream ofs(FP);

	int len = P.totalAA;
	u_int tlen = 0;
	u_int tsize = P.targetsize;
	string ID = "";
	int pos = 0;
	char *pAA = P.Seq;
	float *pD = D;
	u_int *pR = read;
	float* pRatio = AA;
	int i = 0; int j = 0; int k = 0;
	for (i = 0; i < tsize; ++i) {
		tlen = P.Seqlength[i];
		ID = P.filename[i].substr(1, P.filename[i].length() - 4);
		for (j = 0; j < tlen; ++j, ++pAA, ++pD, ++pR, ++pos) {
			ofs << ID.c_str() << "," << j << "," << AAlist[P.Seq[pos]] << "," << *pR << "," << *pD << ",";
			for (k = 0; k < 21; ++k, ++pRatio) {
				ofs << *pRatio;
			}
			ofs << "\n";
		}

	}
	ofs.close();
	*/

}
inline void checknun(float *d, size_t len) {
	for (int i = 0; i < len; ++i) {
		if (isnan(d[i]))
			d[i] = 0;
	}
}

inline void Export_result(float *D, float *AA, float *read, const string &export_d, const string barcode, const PV &P) {
	Directory_check(export_d);

	string FP = export_d + "\\" + barcode + ".csv";

	FILE *stream;

	fopen_s(&stream, (FP).c_str(), "a");
	if (stream == NULL) {
		printf("%s file not open!\n", (FP).c_str());
	}
	else {
		char *buf = new char[1024 * 1024 * 1024];
		setvbuf(stream, buf, _IOFBF, 1024 * 1024 * 1024);
		int len = P.totalAA;
		u_int tlen = 0;
		u_int tsize = P.targetsize;
		string ID = "";
		int pos = 0;
		char *pAA = P.Seq;
		float *pD = D;
		float *pR = read;
		float* pRatio = AA;
		int i = 0; int j = 0; int k = 0;
		for (i = 0; i < tsize; ++i) {
			tlen = P.Seqlength[i];
			ID = P.filename[i].substr(1, P.filename[i].length() - 4);
			for (j = 0; j < tlen; ++j, ++pAA, ++pD, ++pR, ++pos) {
				//fprintf(stream, "%s,%d,%c,%f,%f", ID.c_str(), j, AAlist[P.Seq[pos]], *pR, *pD);
				fprintf(stream, "%s,%d,%c,%f,%f", ID.c_str(), j, AAlist[P.Seq[pos]], *pR, *pD);
				for (k = 0; k < 21; ++k, ++pRatio) {
					fprintf(stream, ",%f", *pRatio);
				}
				fprintf(stream, "\n");
			}

		}

		fclose(stream);
		delete[] buf;
	}
}
inline void Export_result2(float *D, float *AA, float *read, const string &export_d, const string barcode, const PV &P, double *RR) {
	Directory_check(export_d);

	string FP = export_d + "\\" + barcode + ".csv";

	FILE *stream;

	fopen_s(&stream, (FP).c_str(), "a");
	if (stream == NULL) {
		printf("%s file not open!\n", (FP).c_str());
	}
	else {
		char *buf = new char[1024 * 1024 * 1024];
		setvbuf(stream, buf, _IOFBF, 1024 * 1024 * 1024);
		int len = P.totalAA;
		u_int tlen = 0;
		u_int tsize = P.targetsize;
		string ID = "";
		int pos = 0;
		char *pAA = P.Seq;
		float *pD = D;
		float *pR = read;
		double *pRR = RR;
		float* pRatio = AA;
		int i = 0; int j = 0; int k = 0;
		for (i = 0; i < tsize; ++i) {
			tlen = P.Seqlength[i];
			ID = P.filename[i].substr(1, P.filename[i].length() - 4);
			for (j = 0; j < tlen; ++j, ++pAA, ++pD, ++pR, ++pos) {
				//fprintf(stream, "%s,%d,%c,%f,%f", ID.c_str(), j, AAlist[P.Seq[pos]], *pR, *pD);
				fprintf(stream, "%s,%d,%c,%f,%f", ID.c_str(), j, AAlist[P.Seq[pos]], *pR, *pD);
				for (k = 0; k < 12; ++k, ++pRR) {
					fprintf(stream, ",%lf", *pRR);
				}
				for (k = 0; k < 21; ++k, ++pRatio) {
					fprintf(stream, ",%f", *pRatio);
				}
				fprintf(stream, "\n");
			}

		}

		fclose(stream);
		delete[] buf;
	}
	/*
	ofstream ofs(FP);

	int len = P.totalAA;
	u_int tlen = 0;
	u_int tsize = P.targetsize;
	string ID = "";
	int pos = 0;
	char *pAA = P.Seq;
	float *pD = D;
	u_int *pR = read;
	float* pRatio = AA;
	int i = 0; int j = 0; int k = 0;
	for (i = 0; i < tsize; ++i) {
		tlen = P.Seqlength[i];
		ID = P.filename[i].substr(1, P.filename[i].length() - 4);
		for (j = 0; j < tlen; ++j, ++pAA, ++pD, ++pR, ++pos) {
			ofs << ID.c_str() << "," << j << "," << AAlist[P.Seq[pos]] << "," << *pR << "," << *pD << ",";
			for (k = 0; k < 21; ++k, ++pRatio) {
				ofs << *pRatio;
			}
			ofs << "\n";
		}

	}
	ofs.close();
	*/

}
inline void Export_result_Q(const string &export_d, const string &fname, int length) {
	Directory_check(export_d);
	string FP = export_d + "/" + fname + ".csv";
	FILE *stream;
	// mtxf.lock();
	// cout << export_d + "/" + fname + ".csv" << endl;
	fopen_s(&stream, (FP).c_str(), "w");
	if (stream == NULL) {
		printf("%s file not open!\n", (FP).c_str());
	}
	else {
		// char buf[512 * 512];
		// setvbuf(stream, buf, _IOFBF, sizeof(buf));
		int pro = 0;
		for (const auto &i : queue) {
			if (queflag[pro])
				for (const auto &j : i) {
					for (const string &k : j) {
						fprintf(stream, "%s,", k);
					}
					fprintf(stream, "\n");
				}
			++pro;
		}

		fclose(stream);
	}
	// mtxf.unlock();
	// delete[] buf;
}
inline char* import_target(const string &FP, int len) {
	FILE *stream;
	char* seq = new char[len];
	cout << "\rimport seq :" << FP << "\t\t";
	//cout << "length: " << len << endl;
	int ret = fopen_s(&stream, (FP).c_str(), "rb");
	if (ret == 0) {
		// cout << result.scoresize <<endl;
		int iret = fread(seq, sizeof(char), len, stream);
		fclose(stream);
		if (iret != len) cout << "fread error" << iret << endl;
	}
	else {
		cout << "\rimport seq :" << FP << "error" << endl;
	}
	return seq;
}
inline void Check_P_func(const string &FP, const vector<string> &fname, vector<int> &len, vector<string> &fn, PV &P) {

	FILE *stream;
	int i = 0;
	while (1) {
		mtx.lock();
		i = task;
		task++;
		if (i >= fname.size())
		{
			mtx.unlock();
			break;
		}
		cout << i << "\rCheck : " << FP << "\\" << fname[i] << "\t\t";
		mtx.unlock();
		//cout << "\rCheck : " << FP << "\\" << fname[i] << "\t\t";
		fopen_s(&stream, (FP + "\\" + fname[i]).c_str(), "rb");
		int buf[2] = {};
		int iret = fread(buf, sizeof(int), 2, stream);
		if (iret != 0) {
			len[i] = buf[0];
			fn[i] = fname[i];
			if (i == 0)
				P.scoresize = (int)buf[1];
			fclose(stream);
		}
		//	cout << i << "\rCheck end : " << FP << "\\" << fname[i] << "\n\t";
		//	Sleep(1000);
	}
}
inline void import_Seq(const string &FP, vector<int> &len, vector<string> &fn, char *seq) {
	FILE *stream;
	int i = 0;
	while (1) {
		mtx.lock();
		i = task;
		task++;
		if (i >= fn.size())
		{
			mtx.unlock();
			break;
		}
		cout << "\rimport seq :" << fn[i].substr(1) << "\t\t";
		mtx.unlock();
		size_t pos = 0;
		for (int n = 0; n < i; ++n) {
			//cout << len[n] << endl;
			pos += len[n];
		}
		FILE *stream;
		char* s = new char[len[i]];

		int ret = fopen_s(&stream, (FP + "\\Seq" + fn[i].substr(1)).c_str(), "rb");
		if (ret == 0) {
			// cout << result.scoresize <<endl;
			int iret = fread(s, sizeof(char), len[i], stream);
			fclose(stream);
			if (iret != len[i]) cout << "fread error" << iret << endl;
		}
		else {
			cout << "\rimport seq :" << FP << "error" << endl;
		}
		char *pseq = &seq[pos];
		for (int n = 0; n < len[i]; ++n, ++s, ++pseq) {
			*pseq = *s;
			//cout << AAlist[*s] << endl;
		}
		//mtx.unlock();
	}

}
inline void Import_P_func(const string &FP, vector<int> &len, vector<string> &fn, PV &P, size_t totallength, float *buf) {
	FILE *stream;
	int i = 0;
	while (1) {
		mtx.lock();
		i = task;
		task++;
		if (i >= fn.size())
		{
			mtx.unlock();
			break;
		}
		cout << "\rImport : " << FP << "\\" << fn[i] << "\t\t";
		mtx.unlock();
		//cout << len[i] << endl;
		double* b = new double[len[i] * P.scoresize];
		fopen_s(&stream, (FP + "\\" + fn[i]).c_str(), "rb");
		_fseeki64(stream, 2 * sizeof(int), SEEK_SET);
		size_t pos = 0;
		for (int n = 0; n < i; ++n) {
			pos += len[n];
		}
		//size_t pos = std::accumulate(len.begin(), len.begin() + i - 1, 0);
		int ret = fread(b, sizeof(double), len[i] * P.scoresize, stream);
		fclose(stream);
		if (ret == len[i] * P.scoresize) {
			for (int j = 0; j < len[i] * P.scoresize; ++j) {
				buf[pos*P.scoresize + j] = (float)(b[j] * FloatSHIFT);//shift 
			}
		}
		else {
			cout << "Import P error: " << (FP + "\\" + fn[i]) << endl;
		}
		delete[] b;
		//cout << "\rImport end : " << FP << "\\" << fn[i] << "\t\t";
		//mtx.unlock();
	}

}

PV Import_P_mt(const string &FP, const vector<string> &fname) {
	PV P;
	vector<thread> threads(10);
	vector<int> len(fname.size());
	vector<string> fn(fname.size());
	task = 0;
	for (int g = 0; g < threads.size(); ++g) {
		threads[g] = thread([&] {Check_P_func(FP, fname, len, fn, P); });
	}
	for (auto &t : threads) {
		t.join();
	}
	cout << "\nCheck_P Finished" << endl;
	size_t totallength = accumulate(len.begin(), len.end(), 0);
	cout << "total length :" << totallength << " : Score size " << P.scoresize << endl;
	char *seq = new char[totallength];
	float *AA = new float[totallength * 21];
	float *Plist = new float[totallength * P.scoresize];
	string *filename = new string[fname.size()];
	int *seqlen = new int[fname.size()];
	task = 0;
	cout << "\nImport_P start" << endl;
	//COUT_VEC_String(fn);
	cout << "Plist size: " << totallength * P.scoresize << endl;
	for (int g = 0; g < threads.size(); ++g) {
		threads[g] = thread([&] {Import_P_func(FP, len, fn, P, totallength, Plist); });
	}
	for (auto &t : threads) {
		t.join();
	}

	cout << "\nImport_P Finished" << endl;
	task = 0;
	for (int g = 0; g < threads.size(); ++g) {
		threads[g] = thread([&] {import_Seq(FP, len, fn, seq); });
	}
	for (auto &t : threads) {
		t.join();
	}

	for (int i = 0; i < fn.size(); ++i) {
		filename[i] = fn[i];
		seqlen[i] = len[i];
	}


	P.filename = filename;
	P.Seq = seq;
	P.scorelist = Plist;
	P.Seqlength = seqlen;
	P.targetsize = fn.size();
	P.totalAA = totallength;
	return P;
}

void ReadP_listcondition(vector<int> &v, const string &FP,
	const vector<string> &fname, int &size) {
	FILE *stream;
	int i = 0;
	int s = v.size();
	while (i < s) {
		mtxt.lock();
		i = task;
		task++;
		if (i < s) {
			cout << "\r" << fname[i];
			mtxt.unlock();
		}
		else {
			mtxt.unlock();
			break;
		}

		fopen_s(&stream, (FP + "/" + fname[i]).c_str(), "rb");
		int buf[2] = {};
		int iret = fread(buf, sizeof(int), 2, stream);
		v[i] = buf[0];
		size = buf[1];
		fclose(stream);
	}
}

void Calc_P_LessThan_Score(PV &P, int size) {
	double sum = 0;
	float *pS = P.scorelist;
	float a = 0;
	int slen = P.scoresize;
	//	cout << "\rP convert : " << t << "/" << P.size();
	for (int i = 0; i < P.targetsize; ++i) {
		int len = P.Seqlength[i];
		for (int j = 0; j < len; ++j) {
			sum = 0;
			for (int k = 0; k < slen; ++k, pS++) {
				a = *pS;

				*pS = (float)(FloatSHIFT - sum);
				if (*pS < 0)
					*pS = 0.0;
				sum += a;
			}
		}
	}
}


__global__ void Convertratio(float *d_Scoreratio, float *d_AAratio, int slen, size_t size) {

	size_t idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx > size) return;
	int i = 0;
	int j = 0;
	float a = 1.0;
	float b = FloatSHIFT;
	float sum = 0.0;
	for (int i = 0; i < slen; ++i) {
		sum += d_Scoreratio[idx * slen + i];
	}
	if (sum == 0.0) {
		for (int i = 0; i < slen; ++i)
			d_Scoreratio[idx * slen + i] = -1;
	}
	else {
		for (int i = 0; i < slen; ++i) {
			a = d_Scoreratio[idx * slen + i];
			d_Scoreratio[idx * slen + i] = b;
			b = b - (a / sum)*FloatSHIFT;
			//printf("%f, ",b);
		}
	}

	sum = 0.0;
	for (int i = 0; i < 21; ++i) {
		sum += d_AAratio[idx * 21 + i];
	}
	if (sum == 0) {
		for (int i = 0; i < 21; ++i)
			d_AAratio[idx * 21 + i] = 0;
	}
	else {
		for (int i = 0; i < 21; ++i) {
			//printf("%f,%f\n", d_AAratio[idx * 21 + i], sum);
			d_AAratio[idx * 21 + i] = d_AAratio[idx * 21 + i] / sum;
		}
	}
}
__global__ void reset_AA(float *d_Scoreratio, float *d_AAratio, float *d_read_TH, int slen, size_t threads) {
	size_t idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx > threads) return;
	for (int i = 0; i < 21; ++i)
		d_AAratio[idx * 21 + i] = 0;
	for (int i = 0; i < slen; ++i)
		d_Scoreratio[idx * slen + i] = 0;
	d_read_TH[idx] = 0;
}
__global__ void match(char* d_AAMap, char *d_pep, char *d_pro, float *d_Scoreratio, float *d_AAratio, float *P,
	float *d_read_TH, float *d_read, float TH, int slen, size_t pepsize, size_t threads, int plength) {
	size_t idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx > threads) return;
	int i = 0;
	int j = 0;
	size_t it = 0;
	int score = 0;
	for (int n = 0; n < plength; ++n) {
		score = 0;
		d_pep[idx*PEPTIDE_LENGTH];
		for (j = 0; j < PEPTIDE_LENGTH; ++j) {
			score += d_AAMap[d_pep[idx*PEPTIDE_LENGTH + j] * 21 + d_pro[n + j]];
		}
		atomicAdd(&d_Scoreratio[n * slen + score], (d_read[idx]) / FloatSHIFT_INT);
		//atomicAdd(&d_Scoreratio[n * slen + score], 1.0);
		//d_Scoreratio[n * slen + score] += ((float)d_read[idx]) / FloatSHIFT_INT;

		if (P[n*slen + score] < TH) {
			for (int j = 0; j < PEPTIDE_LENGTH; ++j) {
				d_AAratio[(n + j) * 21 + d_pep[idx*PEPTIDE_LENGTH + j]] += (d_read[idx]) / FloatSHIFT_INT;
			}
			atomicAdd(&d_read_TH[n], d_read[idx]);
			//d_read_TH[n] += d_read[idx];
			//printf("%d. %f, %u\n", score, P[idx*slen + score], d_read[i]);
		}
		//printf("%d\n", score);
	}

}
__global__  void ChackSeq(char* s, int size) {
	printf("ChackSeq\n");
	for (int i = 0; i < size; ++i) {
		printf("%d", s[i]);
	}
	printf("\n");
}
__global__  void ChackRead(float* s, int size) {
	printf("ChackRead\n");
	for (int i = 0; i < size; ++i) {
		printf("%f\n", s[i]);
	}
	printf("\n");
}

void Matching(const string &dir_name, const vector<string> &sample_list,
	const string &export_d, const PV P_list, string fname, const Pep_data &PepList, mems &m) {
	cout << "Start Matching " << endl;
	chrono::system_clock::time_point start, end, end1, end2, end3, end4, end5;
	start = std::chrono::system_clock::now();
	size_t maxblockx = 1024;
	size_t maxblocky = 1024;
	size_t maxgridx = 2147483647;
	size_t maxgridy = 65535;
	size_t maxgridz = 65535;
	int blocksize = 32;
	// 10496 core
	// max texture dimension size:(131072)(131072,65536)(16384,16384,16384)
	// max dimension size of  a thread block: (1024x1024x64)
	// max dimension size of a grid size : (2147483647,65535,65535)
	size_t threads = PepList.totalPeps;
	dim3 block(blocksize, 1, 1);
	dim3 grid(threads / block.x + 1, 1, 1);
	size_t i = 0;
	double duration = 0;
	double totaltime = 0;
	start = std::chrono::system_clock::now();
	//cout << fname << endl;
	//if(fname=="134")
	//ChackRead << <1,1 >> > ( m.d_read, PepList.totalPeps);
	match << <grid, block >> > (m.d_AAmap, m.d_pep, m.d_pro, m.d_Scoreratio, m.d_AAratio, m.d_P_list, m.d_read_TH, m.d_read, (float)OPs.TH, P_list.scoresize, PepList.totalPeps, threads, P_list.totalAA);
	hipDeviceSynchronize();
	//	cout << "Calc matching finished" << endl;
	//if (fname == "134")
		//ChackRead << <1, 1 >> > (m.d_read_TH, P_list.totalAA);
	size_t threads_AA = P_list.totalAA;
	dim3 block_AA(1024, 1, 1);
	dim3 grid_AA(threads_AA / block_AA.x + 1, 1, 1);
	Convertratio << <grid_AA, 1024 >> > (m.d_Scoreratio, m.d_AAratio, P_list.scoresize, P_list.totalAA);
	hipDeviceSynchronize();
	cout << "end Convertratio " << endl;
	hipMemcpy(m.h_result, m.d_Scoreratio, P_list.totalAA * P_list.scoresize * sizeof(float), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	hipMemcpy(m.h_AAratio, m.d_AAratio, P_list.totalAA * 21 * sizeof(float), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	hipMemcpy(m.h_read_TH, m.d_read_TH, P_list.totalAA * sizeof(float), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	cout << "end hipMemcpy " << endl;
	end1 = std::chrono::system_clock::now();
	//if (OPs.Q_Export)
	//	Export_Q2(m.h_result, P_list, export_d, fname);
	double matchingtime = static_cast<double>(std::chrono::duration_cast<std::chrono::microseconds>(end1 - start).count());
	cout << "matchingtime:  " << matchingtime / 1000000 << " s" << endl;
	//cout << "CalcDistance" << j << endl;
	//CalcReadRatio(m.h_result, P_list, m.h_Read_Ratio);
	CalcDistance(m.h_result, P_list, m.h_distance);
	end2 = std::chrono::system_clock::now();
	double distancetime = static_cast<double>(std::chrono::duration_cast<std::chrono::microseconds>(end2 - end1).count());
	cout << "distancetime:  " << distancetime / 1000000 << " s" << endl;
	checknun(m.h_distance, P_list.totalAA);
	Export_result(m.h_distance, m.h_AAratio, m.h_read_TH, export_d, fname, P_list);
	end3 = std::chrono::system_clock::now();
	if (OPs.Q_Export)
		Export_Q(m.h_result, P_list, export_d, fname);
	double exporttime = static_cast<double>(std::chrono::duration_cast<std::chrono::microseconds>(end3 - end2).count());
	cout << "exporttime:  " << exporttime / 1000000 << " s" << endl;

	reset_AA << <grid_AA, 1024 >> > (m.d_Scoreratio, m.d_AAratio, m.d_read_TH, P_list.scoresize, P_list.totalAA);
	//hipDeviceSynchronize();
}
void MemFree1(mems &m) {
	hipHostFree(m.h_distance);
	hipHostFree(m.h_AAratio);
	hipHostFree(m.h_Scoreratio);
	hipHostFree(m.h_read_TH);
	hipHostFree(m.h_result);
	hipHostFree(m.h_Read_Ratio);
	hipFree(m.d_pro);
	hipFree(m.d_AAmap);
	hipFree(m.d_P_list);
	hipFree(m.d_read_TH);
	hipFree(m.d_AAratio);
	hipFree(m.d_Scoreratio);

}
void MemFree(mems &m) {
	cout << "MemFree" << endl;
	hipFree(m.d_pep);
	hipFree(m.d_read);

}
void MemRefresh(mems &m, const PV &P_list) {
	cout << "MemRefresh" << endl;
	for (int i = 0; i < P_list.totalAA; ++i) {
		m.h_distance[i] = 0;
		m.h_Read_Ratio[i] = 0;
		m.h_read_TH[i] = 0;
		m.h_result[i] = 0;
		for (int j = 0; j < 21; ++j)
			m.h_AAratio[i * 21 + j] = 0;
		for (int j = 0; j < P_list.scoresize; ++j)
			m.h_Scoreratio[i * P_list.scoresize + j] = 0;
	}
}
void MemPrep1(mems &m, const PV &P_list, size_t &GPU_mem, size_t &Host_mem) {
	cout << " Total Protein length: " << P_list.totalAA << " aa" << endl;
	hipError_t err;
	GPU_mem = P_list.totalAA + 21 * 21 + sizeof(float) * P_list.totalAA * P_list.scoresize +
		sizeof(float) * P_list.totalAA + sizeof(float) * P_list.totalAA * 21 +
		sizeof(float) * P_list.totalAA * P_list.scoresize;

	Host_mem = sizeof(float) *P_list.totalAA + sizeof(float) *P_list.totalAA * 21 + sizeof(float) *P_list.totalAA * P_list.scoresize +
		sizeof(float) *P_list.totalAA +
		sizeof(float) *P_list.totalAA * P_list.scoresize;

	// on host

	CUDA_SAFE_CALL(err = hipHostMalloc(&m.h_distance, sizeof(float) * P_list.totalAA));
	CUDA_SAFE_CALL(err = hipHostMalloc(&m.h_AAratio, sizeof(float) * P_list.totalAA * 21));
	CUDA_SAFE_CALL(err = hipHostMalloc(&m.h_Scoreratio, sizeof(float) * P_list.totalAA * P_list.scoresize));
	CUDA_SAFE_CALL(err = hipHostMalloc(&m.h_read_TH, sizeof(float) * P_list.totalAA));
	CUDA_SAFE_CALL(err = hipHostMalloc(&m.h_result, sizeof(float) * P_list.totalAA * P_list.scoresize));
	CUDA_SAFE_CALL(err = hipHostMalloc(&m.h_Read_Ratio, sizeof(float) * P_list.totalAA * 12));

	MemRefresh(m, P_list);
	// on device
	CUDA_SAFE_CALL(err = hipMalloc(&m.d_pro, P_list.totalAA));
	CUDA_SAFE_CALL(err = hipMalloc(&m.d_AAmap, 21 * 21));
	CUDA_SAFE_CALL(err = hipMalloc(&m.d_read_TH, sizeof(float) * P_list.totalAA));
	CUDA_SAFE_CALL(err = hipMalloc(&m.d_AAratio, sizeof(float) * P_list.totalAA * 21));
	CUDA_SAFE_CALL(err = hipMalloc(&m.d_Scoreratio, sizeof(float) * P_list.totalAA * P_list.scoresize));
	CUDA_SAFE_CALL(err = hipMalloc(&m.d_P_list, sizeof(float) * P_list.totalAA * P_list.scoresize));

	CUDA_SAFE_CALL(err = hipMemcpy(m.d_AAmap, AAMap, 21 * 21, hipMemcpyHostToDevice));  // AAmap (21 * 21)
	CUDA_SAFE_CALL(err = hipMemcpy(m.d_AAratio, m.h_AAratio, P_list.totalAA * 21 * sizeof(float), hipMemcpyHostToDevice));  //
	CUDA_SAFE_CALL(err = hipMemcpy(m.d_Scoreratio, m.h_Scoreratio, P_list.totalAA * P_list.scoresize * sizeof(float), hipMemcpyHostToDevice));  //
	CUDA_SAFE_CALL(err = hipMemcpy(m.d_P_list, P_list.scorelist, P_list.totalAA * P_list.scoresize * sizeof(float), hipMemcpyHostToDevice));  //
	CUDA_SAFE_CALL(err = hipMemcpy(m.d_pro, P_list.Seq, P_list.totalAA, hipMemcpyHostToDevice));  //



}

void MemPrep(mems &m, Pep_data &PepList, size_t &GPU_mem, size_t &Host_mem) {
	cout << "Total peptide num: " << PepList.totalPeps << endl;
	hipError_t err;
	GPU_mem += PepList.peplength + sizeof(float) * PepList.totalPeps + sizeof(float) * PepList.totalPeps;
	cout << "GPU mem: " << GPU_mem / 1000000 << " MB" << endl;
	cout << "Host mem: " << Host_mem / 1000000 << " MB" << endl;

	// on device
	cout << "hipMalloc" << endl;
	char *mem1;
	float *mem2;
	m.d_pep = mem1;
	m.d_read = mem2;
	cout << "d_pep" << endl;
	cout << "malloc: " << PepList.peplength << " byte" << endl;
	int count = 0;
	err = hipMalloc((void **)&m.d_pep, PepList.peplength);
	while (err != hipSuccess) {
		Sleep(1000);
		cout << "malloc: " << PepList.peplength << " byte" << endl;
		err = hipMalloc((void **)&m.d_pep, PepList.peplength);
		count++;
		if (count > 4) {
			exit(err);
		}
	}
	cout << "d_read" << endl;
	err = hipMalloc((void **)&m.d_read, sizeof(float) * PepList.totalPeps);
	while (err != hipSuccess) {
		Sleep(1000);
		cout << "d_read" << endl;
		err = hipMalloc((void **)&m.d_read, sizeof(float) * PepList.totalPeps);
		count++;
		if (count > 4) {
			exit(err);
		}
	}
	cout << "hipMemcpy" << endl;
	CUDA_SAFE_CALL(err = hipMemcpy(m.d_pep, PepList.pep, PepList.peplength, hipMemcpyHostToDevice));  //
	CUDA_SAFE_CALL(err = hipMemcpy(m.d_read, PepList.read, PepList.totalPeps * sizeof(float), hipMemcpyHostToDevice));  //
	//ChackSeq << <1, 1 >> > (m.d_pep, 100);
}
void delPep(Pep_data &PepList) {
	delete[] PepList.read;
	delete[] PepList.pep;
}
void GetGPUProf() {
	hipDeviceProp_t dev;
	hipGetDeviceProperties(&dev, 0);
	cout << "--------------------------------------------------------------" << endl;
	cout << "GPU information" << endl;
	cout << "---------------------------------------------------------------" << endl;
	printf("device %d\n", 0);
	printf(" device name : %s\n", dev.name);
	printf(" total global memory : %d (MB)\n", dev.totalGlobalMem / 1024 / 1024);
	printf(" shared memory / block : %d (KB)\n", dev.sharedMemPerBlock / 1024);
	printf(" register / block : %d\n", dev.regsPerBlock);
	printf(" warp size : %d\n", dev.warpSize);
	printf(" max pitch : %d (B)\n", dev.memPitch);
	printf(" max threads / block : %d\n", dev.maxThreadsPerBlock);
	printf(" max size of each dim. of block : (%d, %d, %d)\n", dev.maxThreadsDim[0], dev.maxThreadsDim[1], dev.maxThreadsDim[2]);
	printf(" max size of each dim. of grid  : (%d, %d, %d)\n", dev.maxGridSize[0], dev.maxGridSize[1], dev.maxGridSize[2]);
	printf(" clock rate : %d (MHz)\n", dev.clockRate / 1000);
	printf(" total constant memory : %d (KB)\n", dev.totalConstMem / 1024);
	printf(" compute capability : %d.%d\n", dev.major, dev.minor);
	printf(" alignment requirement for texture : %d\n", dev.textureAlignment);
	printf(" device overlap : %s\n", (dev.deviceOverlap ? "ok" : "not"));
	printf(" num. of multiprocessors : %d\n", dev.multiProcessorCount);
	printf(" kernel execution timeout : %s\n", (dev.kernelExecTimeoutEnabled ? "on" : "off"));
	printf(" integrated : %s\n", (dev.integrated ? "on" : "off"));
	printf(" host memory mapping : %s\n", (dev.canMapHostMemory ? "on" : "off"));

	printf(" compute mode : ");
	if (dev.computeMode == hipComputeModeDefault) printf("default mode (multiple threads can use) \n");
	else if (dev.computeMode == hipComputeModeExclusive) printf("exclusive mode (only one thread will be able to use)\n");
	else if (dev.computeMode == hipComputeModeProhibited) printf("prohibited mode (no threads can use)\n");
	cout << "---------------------------------------------------------------" << endl;
}

int main(int argc, char *argv[]) {
	clock_t start = clock();
	OPs.threads = thread::hardware_concurrency() - 1;
	try {
		for (int i = 0; i < argc; ++i) {
			cout << i << ": " << argv[i] << " : ";
			if (strstr(argv[i], "--pep") != NULL) {
				++i;
				OPs.FPr = argv[i];
				cout << argv[i];
			}
			if (strstr(argv[i], "--thread") != NULL) {
				++i;
				OPs.threads = stoi(argv[i]);
				cout << argv[i];
			}
			if (strstr(argv[i], "--export") != NULL) {
				++i;
				OPs.FPw = argv[i];
				cout << argv[i];
			}
			if (strstr(argv[i], "--P_map") != NULL) {
				++i;
				OPs.P_map = argv[i];
				cout << argv[i];
			}
			if (strstr(argv[i], "--target") != NULL) {
				++i;
				OPs.Target = argv[i];
				cout << argv[i];
			}
			if (strstr(argv[i], "--Plistexport") != NULL) {
				++i;
				OPs.Plistexport = argv[i];
				cout << argv[i];
			}
			if (strstr(argv[i], "--readlimit") != NULL) {
				++i;
				OPs.PepLimit = stoi(argv[i]);
				std::cout << argv[i];
			}

			if (strstr(argv[i], "--Calc") != NULL) {
				++i;
				std::cout << argv[i];
				if (strstr(argv[i], "read") != NULL) {
					OPs.Calc = 0;
					cout << "mode read";
				}
				else {
					OPs.Calc = 1;
					cout << "mode num";
				}
			}
			if (strstr(argv[i], "--QExport") != NULL) {
				OPs.Q_Export = 1;

			}
			if (strstr(argv[i], "--PExport") != NULL) {
				OPs.P_Export = 1;

			}
			if (strstr(argv[i], "--TH") != NULL) {
				++i;
				OPs.TH = stod(argv[i])*FloatSHIFT;
				cout << OPs.TH;
			}
			if (strstr(argv[i], "--Distance_function") != NULL) {
				++i;
				OPs.Distance_function = stod(argv[i]);
				cout << OPs.Distance_function;
			}
			if (strstr(argv[i], "--Start") != NULL) {
				++i;
				OPs.Start = argv[i];
				cout << OPs.Start;
			}
			cout << endl;
		}
	}
	catch (exception &e) {
		cout << "--pep <*> \t: import directory/n";
		cout << "--export <*> \t: export directory\n";
		cout << "--P_map <*>\t: P map file path\n";
		cout << "-- sim_table <*>\t : AA similarity table\n";
		cout << "--target <*>\t: target protein direcroty\n";
		cout << "--thread <*>\t : thread size(default cours - 2)\n";
		cout << "--Plistexport <*>\t : yes(export P list, default <no> )\n";
		cout << "--readlimit <*>\t : default 0\n";
		cout << "--TH <*>\t : Threshold of cluster (default 0.001)\n";
		cout << "--Calc <read or num>\t : Use NGS read or kinds of peptide "
			"(default read)\n";
		cout << "--Distance_function <*>\t : 1: KL, 2:KL_like, 3:KL_plus\n";
		cout << "--QExport <*>\t : export Q list\n";


		return 0;
	}
	Directory_check(OPs.FPw);
	Copyfile((OPs.P_map + "\\Protein_ID_list.txt").c_str(), (OPs.FPw + "\\Protein_ID_list.txt").c_str());
	vector<string> sample_list = Get_File_path_in_dir(OPs.FPr, "csv");
	samplesize = sample_list.size();
	Load_ScoreTable_Limited2(OPs.P_map + "\\sim_table.csv", AAMap, 0, 11);
	cout << OPs.P_map << endl;
	vector<string> file_list = Get_File_path_in_dir(OPs.P_map, "bin");
	vector<string> P_file_list;
	vector<string> S_file_list;
	GetGPUProf();
	for (int n = 0; n < file_list.size(); ++n) {
		if (file_list[n][0] == 'P')
			P_file_list.push_back(file_list[n]);
		else if (file_list[n].substr(0, 3) == "Seq")
			S_file_list.push_back(file_list[n]);
	}
	sort(P_file_list.begin(), P_file_list.end());
	sort(S_file_list.begin(), S_file_list.end());
	cout << "Import P_file\t\t\t" << endl;
	int targetsize = P_file_list.size();
	int length = 0;

	PV P_list = Import_P_mt(OPs.P_map, P_file_list);

	Calc_P_LessThan_Score(P_list, targetsize);
	cout << "\nImport P_file Finsihed\t\t\t" << endl;
	/*for (int n = 0; n < 1000; ++n) {
		for (int m = 0; m < P_list.scoresize; ++m) {
			printf("%f, ", P_list.scorelist[n * P_list.scoresize + m]);
		}
		cout << endl;
	}*/
	//for (int i = 0; i < P_list.totalAA; ++i)
	//	cout << AAlist[P_list.Seq[i]];



	
	for (int i=stoi(OPs.Start); i < samplesize; ++i) {
		cout << "**********************************************" << endl;
		mems memory;
		size_t GPUmem, Hostmem;
		MemPrep1(memory, P_list, GPUmem, Hostmem);
		// queue = new Qdata[targetsize];
		Pep_data PepList = Import_Peptide(OPs.FPr + "\\" + sample_list[i]);
		size_t tRead = 0;
		for (int n = 0; n < PepList.totalPeps; ++n) {
			tRead += PepList.read[n];
		}
		P_list.totalread = tRead;
		string fname = ExtractPathWithoutExt(sample_list[i]);

		cout << "total read: " << P_list.totalread << "read " << endl;
		cout << "total peptide " << PepList.totalPeps << " peps " << endl;

		if (PepList.totalPeps < 2) {
			delPep(PepList);
			continue;
		}
		MemPrep(memory, PepList, GPUmem, Hostmem);
		MemRefresh(memory, P_list);

		Export_result_header(OPs.FPw, fname);

		Matching(OPs.FPr, sample_list, OPs.FPw, P_list, fname, PepList, memory);
		//hipDeviceSynchronize();
		delPep(PepList);
		MemFree(memory);
		MemFree1(memory);
	}
	if (OPs.P_Export == 1)
		Export_P(P_list, OPs.FPw);
	//hipDeviceReset();
	cout << "***************************************" << endl << endl;
	cout << " Finished " << endl << endl;
	clock_t end = clock();
	cout << "Total: " << difftime(end, start) / 60 / 1000 << "min\n";
	cout << "************************" << endl << endl;
}
